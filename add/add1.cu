
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

/*********************************************
* @author: Zhixiang Wang                     *
* @brief: example for large 1D matrix add    *
* @date: 2023/8/1                            *
*********************************************/

struct Matrix {
    int x;
    int y;
    int z;
};

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;
__global__ void add(const double *x, const double *y, double *z, const int N);
void check(const double *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *host_x = (double*)malloc(M);
    double *host_y = (double*)malloc(M);
    double *host_z = (double*)malloc(M);

    for(int n = 0; n < N; ++n)
    {
        host_x[n] = a;
        host_y[n] = b;
    }

    double *dev_x, *dev_y, *dev_z;
    hipMalloc((void**)&dev_x, M);
    hipMalloc((void**)&dev_y, M);
    hipMalloc((void**)&dev_z, M);

    hipMemcpy(dev_x, host_x, M, hipMemcpyHostToDevice);
    hipMemcpy(dev_y, host_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = (N / block_size) + 1;
    add<<<grid_size, block_size>>>(dev_x, dev_y, dev_z, N);

    hipMemcpy(host_z, dev_z, M, hipMemcpyDeviceToHost);
    check(host_z, N);

    free(host_x);
    free(host_y);
    free(host_z);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_z);
}

__global__ void add(const double *x, const double *y, double *z, const int N)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < N)
    {
        z[idx] = x[idx] +y[idx];
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for(int n = 0; n < N; ++n)
    {
        if(fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }

    printf("%s\n", has_error ? "has errors" : " no errors");
}
