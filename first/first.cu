#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

__global__ void vec_add(int *a, int *b)
{
    int i = blockIdx.x;
    a[i] += b[i];
}

int main()
{
    const int N = 100;
    // ---- allocate the memory ---- //
    int *a, *b;
    hipMallocManaged(&a, N*sizeof(int));
    hipMallocManaged(&b, N*sizeof(int));
    // ---- initialize a,b ---- //
    for(int i=0;i<N;i++)
    {
        a[i] = i;
        b[i] = 2*i;
    }
    // ---- add b to a ---- //
    vec_add<<<N,1>>>(a,b);
    hipDeviceSynchronize();
    // ---- display a ---- //
    for(int i=0;i<N;i++)
    {
        cout << a[i] << endl;
    }
}