#include "hip/hip_runtime.h"
#include "../error.cuh"
#include <stdio.h>

/****************************************************************************
* @author: Zhixiang Wang                                                    *
* @brief: Transpose a matrix                                                *
* @date: 2023/8/2                                                           *
****************************************************************************/

const int NUM_REPEATS = 10;
const int TILE_DIM = 32;

void timing(const int *dev_a, int *dev_b, const int N, const int task);
__global__ void copy(const int *A, int *B, const int N);
__global__ void transpose1 (const int *A, int * B, const int N);
__global__ void transpose2(const int *A, int * B, const int N);
__global__ void transpose3(const int *A, int * B, const int N);
void print_matrix(const int N, const int *A);

int main(int argc, char **argv)
{
    // if (argc != 2)
    // {
    //     printf("usage: %s N\n", argv[0]);
    //     exit(1);
    // }
    // const int N = atoi(argv[1]);
    const int N = 10;

    const int N2 = N * N;
    const int M = sizeof(int) * N2;
    int *h_A = (int *) malloc(M);
    int *h_B = (int *) malloc(M);
    
    for (int n = 0; n < N2; ++n)
    {
        h_A[n] = n;
    }
    int *d_A, *d_B;
    CHECK(hipMalloc(&d_A, M));
    CHECK(hipMalloc(&d_B, M));
    CHECK(hipMemcpy(d_A, h_A, M, hipMemcpyHostToDevice));

    printf("\ncopy:\n");
    timing(d_A, d_B, N, 0);
    printf("\ntranspose with coalesced read:\n");
    timing(d_A, d_B, N, 1);
    printf("\ntranspose with coalesced write:\n");
    timing(d_A, d_B, N, 2);
    printf("\ntranspose with coalesced write and __ldg read:\n");
    timing(d_A, d_B, N, 3);

    CHECK(hipMemcpy(h_B, d_B, M, hipMemcpyDeviceToHost));
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB =\n");
        print_matrix(N, h_B);
    }

    free(h_A);
    free(h_B);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    return 0;
}

void timing(const int *d_A, int *d_B, const int N, const int task)
{
    const int grid_size_x = (N + TILE_DIM -1) / TILE_DIM;
    const dim3 block_size(TILE_DIM, TILE_DIM);
    const dim3 grid_size(grid_size_x, grid_size_x);

    float t_sum = 0;
    float t2_sum = 0;

    for(int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        switch (task)
        {
            case 0:
                copy<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 1:
                transpose1<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 2:
                transpose2<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 3:
                transpose3<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            default:
                printf("Error: wrong task\n");
                exit(1);
                break;
        }
        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }
    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
}

__global__ void copy(const int *A, int *B, const int N)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(x < N && y < N)
    {
        B[x * N + y] = A[x * N + y];
    }
}

__global__ void transpose1(const int *A, int * B, const int N)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(x < N && y < N)
    {
        B[x * N + y] = A[y * N + x];
    }
}

__global__ void transpose2(const int *A, int * B, const int N)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(x < N && y < N)
    {
        B[y * N + x] = A[x * N + y];
    }
}

__global__ void transpose3(const int *A, int * B, const int N)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(x < N && y < N)
    {
        B[y * N + x] = __ldg(&A[x * N + y]);
    }
}

void print_matrix(const int N, const int *A)
{
    for(int y = 0; y < N; ++y)
    {
        for(int x = 0; x < N; ++x)
        {
            printf("%d\t", A[y * N + x]);
        }
        printf("\n");
    }
}
